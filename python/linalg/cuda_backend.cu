#include "cuda_backend.hpp"

#include <linalg/linalg.hpp>
#include <sstream>

void initialise_cuda_backend(py::module& m)
{
    using namespace linalg;

    using size_type = cuda_backend::size_type;

    py::class_<cuda_environment>(m, "cuda_environment")
        .def(py::init(), "Default construct an empty cuda environment.")
        .def(py::init<int, int>(), R"mydelim(
            Construct a cuda environment specifying the device id and number of streams

            :Parameters:    - **device_id** (int) - The cuda device index
                            - **nstreams** (int) - The number of cuda streams to use
            )mydelim")        
        .def("init", &cuda_environment::init, py::arg(), py::arg("nstreams")=1, R"mydelim(
            Construct a cuda environment specifying the device id and number of streams

            :Parameters:    - **device_id** (int) - The cuda device index
                            - **nstreams** (int, optional) - The number of cuda streams to use (Default: 1)
            )mydelim")        
        .def("destroy", &cuda_environment::destroy, R"mydelim(
            Destroys the cuda environment object deallocating any internal memory.
            )mydelim")    
         .def("is_initialised", &cuda_environment::is_initialised, R"mydelim(
            :returns: Whether or not the cuda_environment object has been successfully initialised.
            "rtype: bool
            )mydelim")    
        .def_static("number_of_devices", &cuda_environment::number_of_devices, R"mydelim(
            :returns: The number of cuda devices available on the system
            :rtype: int
        )mydelim")
        .def("list_devices", 
            [](const cuda_environment& o)
            {
                std::ostringstream oss;
                o.list_devices(oss);
                return oss.str();
            }, R"mydelim(
            :returns: A string of the cuda_environmen properties
            :rtype: str
            )mydelim")
        .def("__str__", 
            [](const cuda_environment& o)
            {
                std::ostringstream oss;
                oss << o;
                return oss.str();
            }, R"mydelim(
            :returns: A string of the cuda_environmen properties
            :rtype: str
            )mydelim");

    //expose the ttn node class.  This is our core tensor network object.
    py::class_<cuda_backend>(m, "cuda_backend")
        .def_static("environment", &cuda_backend::environment, py::return_value_policy::reference, R"mydelim(
            Access the cuda environment parameters bound to the backend object.
            )mydelim")
        .def_static("initialise", [](size_type device_id, size_type nstreams){cuda_backend::initialise(device_id, nstreams);}, py::arg("device_id")=0, py::arg("nstreams")=1, R"mydelim(
            Initialise cuda backend passing a user defined environment object.

            :param device_id: The device id used for the cuda backend (Default: 0)
            :type device_id: int, optional
            :param nstreams: The maximum number of streams to use (Default: 1)
            :type nstreams: int, optional
            )mydelim")
        .def_static("destroy", &cuda_backend::destroy, R"mydelim(
            Clear the cuda_backend object.   Free any resources allocated.
            )mydelim")
        .def_static("device_properties", 
            []()
            {
                std::ostringstream oss;
                cuda_backend::device_properties(oss);
                return oss.str();
            }, R"mydelim(
            :returns: A string of the cuda device properties
            :rtype: str
            )mydelim");
}

